#include "kernels.h"

__constant__ CoarsePermIntegrationKernelArgs coarse_perm_int_ctx;

callCoarsePermIntegrationKernel(dim3 grid, dim3 block, CoarsePermIntegrationKernelArgs args){
	hipMemcpyToSymbolAsync(HIP_SYMBOL(coarse_perm_int_ctx), args, sizeof(RKKernelArgs), 0, hipMemcpyHostToDevice);
	CoarsePermIntegrationKernel<<<grid, block>>>;
}

CoarsePermIntegrationKernel<<<grid, block>>>;
